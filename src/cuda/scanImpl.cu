#ifndef SCAN_IMPL_CU
#define SCAN_IMPL_CU

#include "scan.cu"
#include "../include/gpuCudaLib.h"
#include "../include/common.h"

static void scanImpl(int *d_input, int rLen, int *d_output, struct statistic * pp)
{
	int len = 2;
	if(rLen < len){
		int *input, *output;
		hipMalloc((void**)&input,len*sizeof(int));
		hipMalloc((void**)&output, len*sizeof(int));
		hipMemset(input, 0, len*sizeof(int));
		hipMemcpy(input, d_input, rLen*sizeof(int), hipMemcpyDeviceToDevice);
		preallocBlockSums(len);
		prescanArray(output, input, len, pp);
		deallocBlockSums();
		hipMemcpy(d_output,output,rLen*sizeof(int),hipMemcpyDeviceToDevice);
		hipFree(input);
		hipFree(output);
		return;
	}else{

            int siDebug = 0;
		//Start timer for scanImpl Step 4.2.1 - preallocBlockSums time
		struct timespec startScanImplS1, endScanImplS1;
		if(siDebug) clock_gettime(CLOCK_REALTIME,&startScanImplS1);

		preallocBlockSums(rLen);

                if(siDebug) {
                    //Stop timer for scanImpl Step 4.2.1 - preallocBlockSums time
                    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
                    clock_gettime(CLOCK_REALTIME, &endScanImplS1);
                    pp->preallocBlockSums_scanImpl_S1 += (endScanImplS1.tv_sec - startScanImplS1.tv_sec)* BILLION + endScanImplS1.tv_nsec - startScanImplS1.tv_nsec;
                }

		//Start timer for scanImpl Step 4.2.2 - prescanArray time
		struct timespec startScanImplS2, endScanImplS2;
		if(siDebug) clock_gettime(CLOCK_REALTIME,&startScanImplS2);

		prescanArray(d_output, d_input, rLen, pp);

                if(siDebug){
                    //Stop timer for scanImpl Step 4.2.2 - prescanArray time
                    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
                    clock_gettime(CLOCK_REALTIME, &endScanImplS2);
                    pp->prescanArray_scanImpl_S2 += (endScanImplS2.tv_sec - startScanImplS2.tv_sec)* BILLION + endScanImplS2.tv_nsec - startScanImplS2.tv_nsec;
                }

		//Start timer for scanImpl 4.2.3 - deallocBlockSums time
		struct timespec startScanImplS3, endScanImplS3;
		if(siDebug) clock_gettime(CLOCK_REALTIME,&startScanImplS3);

		deallocBlockSums();

                if(siDebug){
                    //Stop timer for scanImpl 4.2.3 - deallocBlockSums time
                    CUDA_SAFE_CALL(hipDeviceSynchronize()); //need to wait to ensure correct timing
                    clock_gettime(CLOCK_REALTIME, &endScanImplS3);
                    pp->deallocBlockSums_scanImpl_S3 += (endScanImplS3.tv_sec - startScanImplS3.tv_sec)* BILLION + endScanImplS3.tv_nsec - startScanImplS3.tv_nsec;
                }
	}
}


#endif

